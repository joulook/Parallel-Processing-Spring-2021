﻿#include "hip/hip_runtime.h"

#include <stdio.h>

#include <stdlib.h>
using namespace std;


__global__ void MatricesMultiply(float* x, float* y, float* z, int s) {

	int C = (blockIdx.x * blockDim.x) + threadIdx.x;
	int R = (blockIdx.y * blockDim.y) + threadIdx.y;
	
	if ((R<s) && (C<s)) {
		float result = 0;
		for (int i = 0; i < s; i++) {
			result += x[R * s + i] * y[i * s + C];
		}
		z[R * s + C] = result;
	}
}

int main(void) {
	int n = 6;
	float* a, * b, * c, * a_d, * b_d, * c_d;

	a = (float*)malloc(sizeof(float) * n * n);
	b = (float*)malloc(sizeof(float) * n * n);
	c = (float*)malloc(sizeof(float) * n * n);

	hipSetDevice(0);

	hipMalloc((void**)&a_d, sizeof(float) * n * n);
	hipMalloc((void**)&b_d, sizeof(float) * n * n);
	hipMalloc((void**)&c_d, sizeof(float) * n * n);

	for (int i = 0; i < n * n; i++) {
		a[i] = i;
		b[i] = i+1;
	}

	hipMemcpy(a_d, a, sizeof(float) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(b_d, b, sizeof(float) * n * n, hipMemcpyHostToDevice);

	dim3 dimGrid(std::ceil(n/32.0), std::ceil(n/32.0), 1);
	dim3 dimBlock(32, 32, 1);
	MatricesMultiply <<<dimGrid, dimBlock>>>(a_d, b_d, c_d, n);
	hipDeviceSynchronize();

	hipMemcpy(c, c_d, sizeof(float) * n * n, hipMemcpyDeviceToHost);

	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);

	printf("The Result is\n\n");
	int counter = 0;
	printf("\t | ");
	for (int i = 0; i < n*n; i++){
		if(counter == n){
			printf("\n");
			printf("\t | ");
			printf("%f\t", c[i]);
			counter = 1;
		}else{
			printf("%f\t", c[i]);
			counter++;
		}

	}
	printf("\n\n");

	return 0;
}

